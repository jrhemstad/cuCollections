#include "hip/hip_runtime.h"
#include <cuda/std/atomic>
#include <thrust/pair.h>
#include <thrust/device_vector.h>
#include <iostream>


template <typename F, typename S>
struct alignas(8) my_pair{
    F first;
    S second;

    template <typename Other>
    __host__ __device__
    my_pair& operator=(Other const&){
        printf("assignment\n");
    }
};

template <typename F, typename S>
__host__ __device__
 my_pair<F,S> make_my_pair(F f, S s){
    return my_pair<F,S>{f,s};
}

__global__ void example(cuda::atomic<my_pair<int,int>, cuda::thread_scope_device> * a) {
    auto expected = make_my_pair(0,0);
    a->compare_exchange_strong(expected, make_my_pair(1,1));
}

int main(){
    using pair_type = my_pair<int, int>;
    std::cout << "alignof pair_type: " << alignof(pair_type) << std::endl
              << "size of pair_type: " << sizeof(pair_type) << std::endl;

    using atomic_pair = cuda::atomic<pair_type, cuda::thread_scope_device>;
    thrust::device_vector<atomic_pair> a(1);
    example<<<1,1>>>(a.data().get());
    if(hipSuccess != hipDeviceSynchronize()){
        std::cout << "error\n";
    }
}
